#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

using namespace std;

#define	RADIX 257
#define SIZE 9216
#define BLOCK_SIZE 512
#define GRID_SIZE 64
#define LOOP_NUM 100

//int HashCalc(char *text, int length);
__host__ void hHashCalc(char *text, int length, unsigned int *rehash);
__global__ void gHashCalc(char *text, int *length, unsigned int *rehash);
__device__ void dHashCalc(char *text, int *length, unsigned int *rehash);
// void textHash(char *text, int textlen, int texthas [], int patlen);
__global__ void textHash(char *text, int *textlen, unsigned int *texthas, int *patlen);
void HashSearch(char *text, int textlen, unsigned int texthas [], char *pattern, int patlen, unsigned int pathas, bool flag []);
__global__ void gHashSearch(char *text, int *textlen, unsigned int *texthas, char *pattern, int *patlen, unsigned int *pathas, bool *flag);
void Emphasis(char *text, int textlen, int patlen, bool flag [], int Count);
void InsertChar(char *text, char *shift, bool flag [], bool mem [], int *counter, char *insert);
void ShiftChar(char *text, char *shift1, char *shift2, bool flag [], bool mem1 [], bool mem2 [], int *counter, int inslen, int looptimes);

int main(){
	char text[SIZE * 2], pattern[SIZE];
	string inputtext;
	int textlen[1], patlen[1];
	unsigned int texthas[SIZE * 2] = { 0 }, pathas[1] = { 0 };
	bool FoundFlag[SIZE] = { 0 };
	int FoundCount = 0;
	int i;
//	cout << "*Please input text." << endl;
	getline(cin, inputtext);
	const char *convert = inputtext.c_str();
	strcpy(text, convert);
	textlen[0] = strlen(text);
//	cout << textlen[0] << endl;

	do{
//		cout << endl << "*Please input pattern." << endl;
		getline(cin, inputtext);
		convert = inputtext.c_str();
		strcpy(pattern, convert);
		patlen[0] = strlen(pattern);


		if (textlen[0] < patlen[0])
		{
			cout << "**Search pattern is larger than the text size.**" << endl;
		}
	} while (textlen[0] < patlen[0]);

	//GPU用変数
	char *dText, *dPattern;
	unsigned int *dTexthas, *dPathas;
	int *dTextlen, *dPatlen;
	bool *dFoundFlag;

	hipMalloc((void**) &dText, sizeof(char)*SIZE);
	hipMemcpy(dText, text, sizeof(char)*SIZE, hipMemcpyHostToDevice);
	hipMalloc((void**) &dPattern, sizeof(char)*SIZE);
	hipMemcpy(dPattern, pattern, sizeof(char)*SIZE, hipMemcpyHostToDevice);
	hipMalloc((void**) &dTexthas, sizeof(unsigned int)*SIZE);
	hipMemcpy(dTexthas, texthas, sizeof(unsigned int)*SIZE, hipMemcpyHostToDevice);
	hipMalloc((void**) &dPathas, sizeof(unsigned int)*SIZE);
	hipMemcpy(dPathas, pathas, sizeof(unsigned int)*SIZE, hipMemcpyHostToDevice);
	hipMalloc((void**) &dTextlen, sizeof(int)*SIZE);
	hipMemcpy(dTextlen, textlen, sizeof(int)*SIZE, hipMemcpyHostToDevice);
	hipMalloc((void**) &dPatlen, sizeof(int)*SIZE);
	hipMemcpy(dPatlen, patlen, sizeof(int)*SIZE, hipMemcpyHostToDevice);

	hipMalloc((void**) &dFoundFlag, sizeof(int)*SIZE);
	hipMemcpy(dFoundFlag, FoundFlag, sizeof(bool)*SIZE, hipMemcpyHostToDevice);


	for (int cnt = 1; cnt <= GRID_SIZE * BLOCK_SIZE; cnt++){
		float Time_pathas = 0;
		float Time_memcpy1 = 0;
		float Time_texthas = 0;
		float Time_memcpy2 = 0;
		float Time_sum = 0;
		float Time_HashSearch = 0;

		dim3 grid(GRID_SIZE * BLOCK_SIZE / cnt);
		dim3 block(cnt);
		while (GRID_SIZE * BLOCK_SIZE % cnt != 0)
		{
			cnt++;
			dim3 grid(GRID_SIZE * BLOCK_SIZE / cnt);
			dim3 block(cnt);
		}

		//タイマーの設定
		//		cout << "Calculation start in the GPU." << endl;
		//		cout << "BlockSize\t:\t" << GRID_SIZE * BLOCK_SIZE / cnt << "\nGridSize\t:\t" << cnt << endl;
		cout << "BlockSize," << GRID_SIZE * BLOCK_SIZE / cnt << "\nGridSize," << cnt << endl;
		//	float sum = 0.0f;

		for (int loopcnt = 0; loopcnt < LOOP_NUM; loopcnt++){
/*			if (loopcnt == 50){
				dim3 grid(cnt);
			}
			*/
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			float millseconds = 0.0f, sum = 0.0f;

			//パターンのハッシュ値計算
			hipEventRecord(start, 0);

			gHashCalc <<<grid, block>>> (dPattern, dPatlen, dPathas);
			hipDeviceSynchronize();

			//	/*
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&millseconds, start, stop);
			sum += millseconds;
			Time_pathas += millseconds;
			//		cout << "Time required(pattern hash)\t:\t" << millseconds << " millseconds" << endl;

			hipEventRecord(start, 0);
			//	*/

			hipMemcpy(pathas, dPathas, sizeof(unsigned int), hipMemcpyDeviceToHost);

			//	/*
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&millseconds, start, stop);
			sum += millseconds;
			Time_memcpy1 += millseconds;
			//		cout << "Time required(memcpy)\t:\t" << millseconds << " millseconds" << endl;

			//	*/

			//		cout << endl << "*Pattern Hash(" << pattern << ") = " << pathas[0] << endl << endl;

			//テキストのハッシュ値計算
			hipEventRecord(start, 0);

			textHash <<<grid, block>>> (dText, dTextlen, dTexthas, dPatlen);
			hipDeviceSynchronize();

			//	/*
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&millseconds, start, stop);
			sum += millseconds;
			Time_texthas += millseconds;
			//		cout << "Time required(text hash)\t:\t" << millseconds << " millseconds" << endl;

			hipEventRecord(start, 0);
			//	*/

			hipMemcpy(texthas, dTexthas, sizeof(unsigned int)*SIZE, hipMemcpyDeviceToHost);

			//	/*
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&millseconds, start, stop);
			sum += millseconds;
			Time_memcpy2 += millseconds;
			//		cout << "Time required(memcpy)\t:\t" << millseconds << " millseconds" << endl;
			//	*/

			Time_sum += sum;
			//		cout << "Time required(sum)\t:\t" << sum << " millseconds" << endl;

			//ハッシュ値比較
			//		cout << "*Finding..." << endl;

			hipEventRecord(start, 0);

			HashSearch(text, textlen[0], texthas, pattern, patlen[0], pathas[0], FoundFlag);
			//gHashSearch << <grid, block >> > (dText, dTextlen, dTexthas, dPattern, dPatlen, dPathas, dFoundFlag);
			//hipMemcpy(FoundFlag, dFoundFlag, sizeof(bool)*SIZE, hipMemcpyDeviceToHost);

			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&millseconds, start, stop);
			sum += millseconds;
			Time_HashSearch += millseconds;
			//		cout << "Time required(HashSearch)\t:\t" << millseconds << " millseconds" << endl;

		}
		cout << "Time required(pattern hash)," << Time_pathas / LOOP_NUM << endl;
		cout << "Time required(memcpy1)," << Time_memcpy1 / LOOP_NUM << endl;
		cout << "Time required(text hash)," << Time_texthas / LOOP_NUM << endl;
		cout << "Time required(memcpy2)," << Time_memcpy2 / LOOP_NUM << endl;
		cout << "Time required(sum)," << Time_sum / LOOP_NUM << endl;
		cout << "Time required(HashSearch)," << Time_HashSearch / LOOP_NUM << endl;



	}
/*
	for (i = 0; i < textlen[0]; i++){
		//cout << "*Text Hash(";
		//for (int j = 0; j < patlen[0]; j++) cout << text[i + j];
		//cout << ") = " << texthas[i] << endl;
		if (FoundFlag[i] == true)	FoundCount++;
	}
	cout << "*FoundCount = " << FoundCount << endl;
	if (FoundCount != 0)
	{
		Emphasis(text, textlen[0], patlen[0], FoundFlag, FoundCount);
		cout << endl << "**Found!!**" << endl << text << endl;
	}
	else
	{
		cout << endl << "**Not found**" << endl;
	}

*/
	
	hipFree(dText);
	hipFree(dPattern);
	hipFree(dTexthas);


	return 0;
}

__host__ void hHashCalc(char *text, int length, unsigned int *rehash)
{
	int scan_idx;
	*rehash = 0;

	for (scan_idx = 0; scan_idx < length; scan_idx++)
	{
		//		rehash = rehash * RADIX + text[i];
		//		rehash += (pow(RADIX, (double)i)) * text[i];
		*rehash += ((scan_idx + 1) * RADIX) * text[scan_idx];
	}

	/*
	cout << "Hash(";
	for(i = 0; i < length; i++) cout << text[i];
	cout << ") = " << rehash << endl;
	*/

}

__global__ void gHashCalc(char *text, int *length, unsigned int *rehash)
{
	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int scan_idx;

	rehash[col_idx] = 0;

	for (scan_idx = 0; scan_idx < *length; scan_idx++)
	{
		rehash[col_idx] += ((scan_idx + 1) * RADIX) * text[col_idx + scan_idx];
		//		*rehash += ((scan_idx + 1) * RADIX) * text[scan_idx];
		//__syncthreads();
	}
	//__syncthreads();
}

__device__ void dHashCalc(char *text, int *length, unsigned int *rehash)
{
	//	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int scan_idx;

	*rehash = 0;

	for (scan_idx = 0; scan_idx < *length; scan_idx++)
	{
		//		*rehash += ((scan_idx + 1) * RADIX) * text[col_idx + scan_idx];
		*rehash += ((scan_idx + 1) * RADIX) * text[scan_idx];
		__syncthreads();
	}

	/*
	cout << "Hash(";
	for(i = 0; i < length; i++) cout << text[i];
	cout << ") = " << rehash << endl;
	*/
}

__global__ void textHash(char *text, int *textlen, unsigned int *texthas, int *patlen)
{
	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int scan_idx;

	texthas[col_idx] = 0;
	for (scan_idx = 0; scan_idx < *patlen; scan_idx++){
		texthas[col_idx] += ((scan_idx + 1) * RADIX) * text[col_idx + scan_idx];
		//__syncthreads();
	}
	//__syncthreads();
}


__global__ void gHashSearch(char *text, int *textlen, unsigned int *texthas, char *pattern, int *patlen, unsigned int *pathas, bool *flag)
{
	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (*pathas == texthas[col_idx])
	{
	int scan_idx = 0;
		do{
			if (text[col_idx + scan_idx] != pattern[scan_idx])	break;
		} while (++scan_idx < *patlen);

		if (scan_idx == *patlen)
		{
			flag[col_idx] = true;
		}
	}
}


void HashSearch(char *text, int textlen, unsigned int texthas [], char *pattern, int patlen, unsigned int pathas, bool flag [])
{
	int i, j;

	for (i = 0; i < textlen - patlen + 1; i++)
	{
		if (pathas == texthas[i])
		{
			//cout << "Found the same hash!" << endl;
	/*		cout << "Text Hash(";
			for (j = 0; j < patlen; j++) cout << text[i + j];
			cout << ") = " << texthas[i] << endl;
	*/
			j = 0;
			do{
				if (text[i + j] != pattern[j])	break;
			} while (++j < patlen);

			if (j == patlen)
			{
				flag[i] = true;
			}
		}
	}
}

void Emphasis(char *text, int textlen, int patlen, bool flag [], int Count)
{
	int i, looptimes;
	char shift1[SIZE], shift2[SIZE];
	bool mem1[SIZE * 2], mem2[SIZE * 2];
	char insert1 [] = " << ", insert2 [] = " >> ";
	int inslen = strlen(insert1);

	looptimes = textlen - patlen + (8 * Count);

	for (i = 0; i < textlen - patlen + (8 * Count); i++)
	{
		if (flag[i] == true)
		{
			InsertChar(text, shift1, flag, mem1, &i, insert1);
			ShiftChar(text, shift1, shift2, flag, mem1, mem2, &i, inslen, looptimes);
			i += patlen;

			InsertChar(text, shift1, flag, mem1, &i, insert2);
			ShiftChar(text, shift1, shift2, flag, mem1, mem2, &i, inslen, looptimes);

		}
	}
}

void InsertChar(char *text, char *shift, bool flag [], bool mem [], int *counter, char *insert)
{
	int inslen = strlen(insert), j;
	for (j = 0; j < inslen; j++)
	{
		shift[j] = text[*counter + j];
		mem[j] = flag[*counter + j];
	}
	for (j = 0; j < inslen; j++)
	{
		text[*counter + j] = insert[j];
	}
	*counter += inslen;
}

void ShiftChar(char *text, char *shift1, char *shift2, bool flag [], bool mem1 [], bool mem2 [], int *counter, int inslen, int looptimes)
{
	int j;
	for (j = 0; j < looptimes; j++)
	{
		shift2[j] = text[*counter + j];
		mem2[j] = flag[*counter + j];
		if (j < inslen){
			text[*counter + j] = shift1[j];
			flag[*counter + j] = mem1[j];
		}
		else{
			text[*counter + j] = shift2[j - inslen];
			flag[*counter + j] = mem2[j - inslen];
		}
	}
}

