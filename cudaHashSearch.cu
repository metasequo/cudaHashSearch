#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

using namespace std;

#define	RADIX 257
#define SIZE 4096
#define BLOCK_SIZE 128
#define GRID_SIZE 16

//int HashCalc(char *text, int length);
__host__ void hHashCalc(char *text, int length, unsigned int *rehash);
__global__ void gHashCalc(char *text, int *length, unsigned int *rehash);
__device__ void dHashCalc(char *text, int *length, unsigned int *rehash);
// void textHash(char *text, int textlen, int texthas [], int patlen);
__global__ void textHash(char *text, int *textlen, unsigned int *texthas, int *patlen);
void HashSearch(char *text, int textlen, unsigned int texthas [], char *pattern, int patlen, unsigned int pathas, int flag []);
void Emphasis(char *text, int textlen, int patlen, int flag [], int Count);
void InsertChar(char *text, char *shift, int flag [], int mem [], int *counter, char *insert);
void ShiftChar(char *text, char *shift1, char *shift2, int flag [], int mem1 [], int mem2 [], int *counter, int inslen, int looptimes);

int main(){
	char text[SIZE], pattern[SIZE];
	string inputtext;
	int textlen[1], patlen[1];
	unsigned int texthas[SIZE * 2] = { 0 }, pathas[1] = { 0 };
	int FoundFlag[SIZE] = { 0 }, FoundCount = 0;
	int i;
	cout << "*Please input text." << endl;
	getline(cin, inputtext);
	cout << endl << "*Please input pattern." << endl;
	cin >> pattern;

	const char *convert = inputtext.c_str();
	strcpy(text, convert);

	textlen[0] = strlen(text);
	patlen[0] = strlen(pattern);

	/*
	for(i = 0; i < patlen; i++)
	{
	FoundFlag[i] = HashCalc(&pattern[i], 1);
	cout << "Hash(" << patterin[i] << ") = " << FoundFlag[i] << endl;
	}
	*/

	if (textlen < patlen)
	{
		cout << "**Search pattern is larger than the text size.**" << endl;
	}
	else
	{

		//GPU
		char *dText, *dPattern;
		unsigned int *dTexthas, *dPathas;
		int *dTextlen, *dPatlen;
		//		int *dFoundFlag;

		hipMalloc((void**) &dText, sizeof(char)*SIZE);
		hipMemcpy(dText, text, sizeof(char)*SIZE, hipMemcpyHostToDevice);
		hipMalloc((void**) &dPattern, sizeof(char)*SIZE);
		hipMemcpy(dPattern, pattern, sizeof(char)*SIZE, hipMemcpyHostToDevice);
		hipMalloc((void**) &dTexthas, sizeof(unsigned int)*SIZE);
		hipMemcpy(dTexthas, texthas, sizeof(unsigned int)*SIZE, hipMemcpyHostToDevice);
		hipMalloc((void**) &dPathas, sizeof(unsigned int)*SIZE);
		hipMemcpy(dPathas, pathas, sizeof(unsigned int)*SIZE, hipMemcpyHostToDevice);
		hipMalloc((void**) &dTextlen, sizeof(int)*SIZE);
		hipMemcpy(dTextlen, textlen, sizeof(int)*SIZE, hipMemcpyHostToDevice);
		hipMalloc((void**) &dPatlen, sizeof(int)*SIZE);
		hipMemcpy(dPatlen, patlen, sizeof(int)*SIZE, hipMemcpyHostToDevice);

		//		hipMalloc((void**) &dFoundFlag, textlen);
		//		hipMemcpy(dFoundFlag, FoundFlag, textlen, hipMemcpyHostToDevice);

		dim3 grid(GRID_SIZE);
		dim3 block(BLOCK_SIZE);

		gHashCalc << <grid, block >> >(dPattern, dPatlen, dPathas);
		hipDeviceSynchronize();

		hipMemcpy(pathas, dPathas, sizeof(unsigned int), hipMemcpyDeviceToHost);
		cout << endl << "*Pattern Hash(" << pattern << ") = " << pathas[0] << endl << endl;

		//hHashCalc(pattern, patlen[0], pathas);
		//cout << endl << "*Pattern Hash(" << pattern << ") = " << pathas[0] << endl << endl;

		cout << "*Finding..." << endl;


		textHash << <grid, block >> >(dText, dTextlen, dTexthas, dPatlen);
		hipDeviceSynchronize();

		hipMemcpy(texthas, dTexthas, sizeof(unsigned int)*SIZE, hipMemcpyDeviceToHost);


		HashSearch(text, textlen[0], texthas, pattern, patlen[0], pathas[0], FoundFlag);
		for (i = 0; i < textlen[0]; i++){
			//cout << "*Text Hash(";
			//for (int j = 0; j < patlen[0]; j++) cout << text[i + j];
			//cout << ") = " << texthas[i] << endl;
			if (FoundFlag[i] == 1)	FoundCount++;

		}
		cout << "*FoundCount = " << FoundCount << endl;
		if (FoundCount != 0)
		{
			Emphasis(text, textlen[0], patlen[0], FoundFlag, FoundCount);
			cout << endl << "**Found!!**" << endl << text << endl;
		}
		else
		{
			cout << endl << "**Not found**" << endl;
		}

		hipFree(dText);
		hipFree(dPattern);
		hipFree(dTexthas);

	}



	return 0;
}

__host__ void hHashCalc(char *text, int length, unsigned int *rehash)
{
	int scan_idx;
	*rehash = 0;

	for (scan_idx = 0; scan_idx < length; scan_idx++)
	{
		//		rehash = rehash * RADIX + text[i];
		//		rehash += (pow(RADIX, (double)i)) * text[i];
		*rehash += ((scan_idx + 1) * RADIX) * text[scan_idx];
	}

	/*
	cout << "Hash(";
	for(i = 0; i < length; i++) cout << text[i];
	cout << ") = " << rehash << endl;
	*/

}

__global__ void gHashCalc(char *text, int *length, unsigned int *rehash)
{
	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int scan_idx;

	rehash[col_idx] = 0;

	for (scan_idx = 0; scan_idx < *length; scan_idx++)
	{
		rehash[col_idx] += ((scan_idx + 1) * RADIX) * text[col_idx + scan_idx];
		//		*rehash += ((scan_idx + 1) * RADIX) * text[scan_idx];
		__syncthreads();
	}
	__syncthreads();
}

__device__ void dHashCalc(char *text, int *length, unsigned int *rehash)
{
	//	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int scan_idx;

	*rehash = 0;

	for (scan_idx = 0; scan_idx < *length; scan_idx++)
	{
		//		*rehash += ((scan_idx + 1) * RADIX) * text[col_idx + scan_idx];
		*rehash += ((scan_idx + 1) * RADIX) * text[scan_idx];
		__syncthreads();
	}

	/*
	cout << "Hash(";
	for(i = 0; i < length; i++) cout << text[i];
	cout << ") = " << rehash << endl;
	*/
}

__global__ void textHash(char *text, int *textlen, unsigned int *texthas, int *patlen)
{
	unsigned int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int scan_idx;


	for (scan_idx = 0; scan_idx < *textlen /*- *patlen + 1*/; scan_idx++){
		//		dHashCalc(&text[scan_idx], patlen, &texthas[scan_idx]);
		texthas[col_idx] = 0;
		for (int i = 0; i < *patlen; i++){
			texthas[col_idx] += ((i + 1) * RADIX) * text[col_idx + i];
			__syncthreads();
		}
		__syncthreads();
	}
	__syncthreads();
}

void HashSearch(char *text, int textlen, unsigned int texthas [], char *pattern, int patlen, unsigned int pathas, int flag [])
{
	int i, j;

	for (i = 0; i < textlen - patlen + 1; i++)
	{
		if (pathas == texthas[i])
		{
			cout << "Found the same hash!" << endl;
			cout << "Text Hash(";
			for (j = 0; j < patlen; j++) cout << text[i + j];
			cout << ") = " << texthas[i] << endl;
			j = 0;
			do{
				if (text[i + j] != pattern[j])	break;
			} while (++j < patlen);

			if (j == patlen)
			{
				flag[i] = 1;
			}
		}
	}
}

void Emphasis(char *text, int textlen, int patlen, int flag [], int Count)
{
	int i, looptimes;
	char shift1[SIZE], shift2[SIZE];
	int mem1[SIZE * 2], mem2[SIZE * 2];
	char insert1 [] = " << ", insert2 [] = " >> ";
	int inslen = strlen(insert1);

	looptimes = textlen - patlen + (8 * Count);

	for (i = 0; i < textlen - patlen + (8 * Count); i++)
	{
		if (flag[i] == 1)
		{
			InsertChar(text, shift1, flag, mem1, &i, insert1);
			ShiftChar(text, shift1, shift2, flag, mem1, mem2, &i, inslen, looptimes);
			i += patlen;

			InsertChar(text, shift1, flag, mem1, &i, insert2);
			ShiftChar(text, shift1, shift2, flag, mem1, mem2, &i, inslen, looptimes);

		}
	}
}

void InsertChar(char *text, char *shift, int flag [], int mem [], int *counter, char *insert)
{
	int inslen = strlen(insert), j;
	for (j = 0; j < inslen; j++)
	{
		shift[j] = text[*counter + j];
		mem[j] = flag[*counter + j];
	}
	for (j = 0; j < inslen; j++)
	{
		text[*counter + j] = insert[j];
	}
	*counter += inslen;
}

void ShiftChar(char *text, char *shift1, char *shift2, int flag [], int mem1 [], int mem2 [], int *counter, int inslen, int looptimes)
{
	int j;
	for (j = 0; j < looptimes; j++)
	{
		shift2[j] = text[*counter + j];
		mem2[j] = flag[*counter + j];
		if (j < inslen){
			text[*counter + j] = shift1[j];
			flag[*counter + j] = mem1[j];
		}
		else{
			text[*counter + j] = shift2[j - inslen];
			flag[*counter + j] = mem2[j - inslen];
		}
	}
}

